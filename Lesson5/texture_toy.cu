
#include <hip/hip_runtime.h>
#include <stdio.h>

__constant__ hipTextureObject_t c_texObj;
__constant__ int texSize;

__global__ void interpolateTexture(float * d_out) {
    int idx = blockDim.x*blockIdx.x + threadIdx.x;

    //d_out[idx] = tex1D<float>(c_texObj, idx + 0.5);
    d_out[idx] = tex1D<float>(c_texObj, idx + 0.5);    
    
}

__host__ void showArray(float * h_out, int size){
    printf("Show output:\n");
    for(int i = 0; i < size; i++){
        printf("Index: %d Value: %1.1f\n", i, h_out[i]);
    }
    printf("\n");
}

int main(){
    const int N = 6;

    // copy N to constant memory as texSize
    hipMemcpyToSymbol(HIP_SYMBOL(texSize), &N, sizeof(N));

    float h_data[N] = {1.,3.,2.,4.,5.,3.};

    // allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray_t cuArray;
    hipMallocArray(&cuArray, &channelDesc, N*sizeof(float), 0);    

    // Copy host array to cuda array in device memory
    // We use 2D functions to allocate a 1D array: 1D functions are DEPRECATED!    
    hipMemcpy2DToArray(cuArray, 0, 0, h_data, N*sizeof(float), N*sizeof(float), 1, hipMemcpyHostToDevice);

    // Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    
    // FILTER MODE    
    texDesc.filterMode = hipFilterModeLinear;
    // READ MODE
    texDesc.readMode = hipReadModeElementType;
    // NORMALIZE COORDINATES
    int normalized = 0;
    texDesc.normalizedCoords = normalized;

    // Create texture object
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    //Copy texObj to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(c_texObj), &texObj, sizeof(texObj));

    int blocks = 1;
    int threads = 8;

    // Allocate result of transformation in device memory
    float *d_output;
    hipMalloc(&d_output, blocks*threads*sizeof(float));

    interpolateTexture<<<blocks, threads>>>(d_output);
    hipDeviceSynchronize();

    //printf("%s",cudaGetErrorString(cudaGetLastError()));

    // copy results back from device memory

    float * h_output = new float[blocks*threads];
    hipMemcpy(h_output, d_output, blocks*threads*sizeof(float), hipMemcpyDeviceToHost);
    
    showArray(h_output, blocks*threads);

    hipFree(d_output);
    //Dedicated cudaFree for Arrays!
    hipFreeArray(cuArray);    
    delete[] h_output;

    return 0;
}
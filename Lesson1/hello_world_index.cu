
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloGPU(){
    int index = threadIdx.x;
    printf("Hello world!!! thread Id: %d \n", index);
}

int main() {
    printf("Launch kernel!\n");
    helloGPU<<<1,1000>>>();
    hipDeviceSynchronize();
    return 0;
}

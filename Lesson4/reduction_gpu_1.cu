
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>
#include <chrono>

#define BLOCKSIZE 64
#define GRIDSIZE 128

typedef std::chrono::high_resolution_clock Clock;
typedef std::chrono::time_point<Clock> timePoint;
typedef std::chrono::duration<double, std::milli> msInterval;

__global__ void reduce1(double *x, int N) {
    int tid = blockDim.x*blockIdx.x + threadIdx.x;
    double tsum = 0.;
    for(int k=tid; k<N; k += gridDim.x*blockDim.x) {        
        // store partial sums in first 
        // BLOCKSIZE * GRIDSIZE element of x
        tsum += x[k]; 
    }
    x[tid] = tsum;    
}

int main() {
    hipDeviceReset();    
    int N = 2097152; // Any number, not necessarily a power of 2
    printf("reduce %d elements\n", N);
    double* h_a = new double[N]; // this time we use new, as in C++

    // Allocate device data
    double* d_a;
    hipMalloc((void **) &d_a, sizeof(double)*N);

    // initalize random generator
    std::default_random_engine gen(42);
    std::uniform_real_distribution<double> fran(0.0, 1.0);

    for(unsigned int i = 0; i < N; i++){
        h_a[i] = fran(gen);
        // uncomment to show on screen the content of h_a
        // %a shows the Hexadecimal floating point representation
        // printf("h_a[%d] == %f %a\n", i, h_a[i], h_a[i]);
    }

    // Copy vector to GPU
    hipMemcpy(d_a, h_a, sizeof(double)*N, hipMemcpyHostToDevice);

    timePoint start = Clock::now();

    // Let's reduce it on the CPU
    double sum = 0.;
    for(unsigned int i = 0; i < N; i++) {
        sum += h_a[i];
    }

    timePoint stop = Clock::now();
    msInterval interval = stop - start;    

    printf("sum on CPU: %f %a\n", sum, sum);
    printf("CPU elapsed time: %f ms \n", interval.count());

    start = Clock::now();

    reduce1<<<GRIDSIZE, BLOCKSIZE>>>(d_a, N);
    reduce1<<<1, BLOCKSIZE>>>(d_a, BLOCKSIZE*GRIDSIZE);
    reduce1<<<1,1>>>(d_a, BLOCKSIZE);
    // Synchronize before time measurement to be sure that all threads are done
    hipDeviceSynchronize();   

    stop = Clock::now();
    interval = stop - start;    

    // sum is in d_a[0], let's copy it from the device to the host variable sum
    double sum_gpu;
    hipMemcpy(&sum_gpu, d_a, sizeof(double), hipMemcpyDeviceToHost);
    
    printf("sum on GPU: %f %a\n", sum_gpu, sum_gpu);    
    printf("GPU elapsed time: %f ms \n", interval.count());

    // Clean memory
    hipFree(d_a);
    delete[] h_a; // remember to use delete[] for arrays instead of delete!!
    return 0;
}